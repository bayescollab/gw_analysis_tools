#include "hip/hip_runtime.h"
#include "autocorrelation_cuda.h"
#include "autocorrelation_cuda.hu"
#include "util.h"
#include <iostream>
#include <condition_variable>
#include <thread>
#include <queue>
#include <functional>
#include <mutex>
#include <unistd.h>
#include <threadPool.h>
#include <hipfft/hipfft.h>


/*! \file
 */

GPUplan *plans_global;

/*! \brief Internal function to calculate the autocorrelation for a given lag
 * Customized for the thread pool architecture, with extra arguments because of the way the memory is allocated
 */
__device__ __host__
void auto_corr_internal(double *arr, /**< Input array of data*/
			int length, /**< Length of input array*/
			int lag,  /**< Lag to be used to calculate the correlation*/
			double average,  /**< Average of the array arr*/
			double *corr,  /**< [out] output correlation*/
			int start_id /**< ID of location to start calculation -- input arrary arr is assumed to be contiguous for multiple dimensions*/
			)
{
	double sum = 0;
	for(int i =0; i< (length - lag); i++){
		sum+= (arr[i+lag+start_id] - average ) * ( arr[i+start_id] - average );
	}		
	*corr = sum / (length - lag);
}

/*! \brief Internal function to launch the CUDA kernel for a range of autocorrelations
 * 
 * Correlation function used:
 *
 * rho(lag) = 1 / (length - lag) \sum (arr[i+lag]-average) ( arr[i]- average)
 *
 * target_corr = rho(rho_index)/rho(0) = rho(rho_index)/var
 */
__global__
void auto_corr_internal_kernal(double *arr, /**< Input array of data*/
				int length,  /**< Length of data array*/
				double average, /**< Average of input data*/
				int *rho_index, /**< [out] Index of the lag that results ina correlation ratio target_corr*/
				double target_corr, /**< Target correlation ratio rho(lag)/rho(0) = target_corr*/
				double var, /**< Variance rho(0)*/
				int start_id/**< Starting index to use for the data array arr*/
				)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id < *rho_index){
		double corr;
		auto_corr_internal(arr, length, id, average, &corr, start_id);
		if(corr/var<target_corr) atomicMin(rho_index, id);
	}

}

/*! \brief Write data file for autocorrelation lengths of the data given a data file name, as written by the mcmc_sampler
 */
void write_file_auto_corr_from_data_file_accel(std::string acfile, /**< Filename of the autocorrelation data*/
					std::string chains_file, /**<Filename of the data file for the chains*/
					int dimension, /**< Dimension of the data*/
					int N_steps, /**< Number of steps in the chain*/
					int num_segments,  /**< Number of segments to check the autocorrelation length for each dimension*/
					double target_corr/**< Target correlation ratio to use for the correlation length calculation*/
					)
{
	double **chains = allocate_2D_array(N_steps, dimension);
	read_file(chains_file, chains, N_steps, dimension);
	write_file_auto_corr_from_data_accel(acfile, chains, dimension, 
			N_steps, num_segments, target_corr);	
	deallocate_2D_array(chains,N_steps, dimension);
}

/*! \brief Write data file given output chains, as formatted by the mcmc_sampler
 */
void write_file_auto_corr_from_data_accel(std::string acfile, /**< Output autocorrelation filename */
					double **chains, /**< Chain data from MCMC_sampler*/
					int dimension, /**< Dimension of the data*/
					int N_steps, /**< Number of steps in the chain*/
					int num_segments,  /**< Number of segments to check the autocorrelation length for each dimension*/
					double target_corr/**< Target correlation ratio to use for the correlation length calculation*/
					)
{
	double **autocorr = allocate_2D_array(dimension, num_segments);
	double **autocorrout = allocate_2D_array(dimension+1, num_segments);
	auto_corr_from_data_accel(chains, dimension, N_steps, num_segments, target_corr, autocorr);
	int seg_step = N_steps/ num_segments;
	for(int i =0 ; i < num_segments; i ++)
	{
		autocorrout[0][i] = (i+1) * seg_step;
		for(int j =0; j<dimension; j++){
			autocorrout[j+1][i] = autocorr[j][i];
		}
	}
	write_file(acfile, autocorrout, dimension+1, num_segments);
	deallocate_2D_array(autocorr,dimension, num_segments);
	deallocate_2D_array(autocorrout,dimension+1, num_segments);
}

/*! \brief Find autocorrelation of data at different points in the chain length and output to autocorr
 */
void auto_corr_from_data_accel(double **output, /**< Chain data input*/
				int dimension, /**< Dimension of the data*/
				int N_steps, /**< Number of steps in the data*/
				int num_segments, /**< number of segments to calculate the autocorrelation length*/
				double target_corr, /**< Target correlation ratio*/
				double **autocorr /**<[out] Autocorrelation lengths for the different segments*/
				)
{
	int device_num;
	hipGetDeviceCount(&device_num);
	chains_internal = output;
	target_corr_internal = target_corr;
	chain_length_internal = N_steps;
	num_segments_internal = num_segments;
	dimension_internal = dimension;
	autocorr_internal = autocorr;

	GPUplan plans[device_num];
	
	for(int i = 0 ; i< device_num; i++)
	{
		plans[i].device_id = i;
		allocate_gpu_plan(&plans[i],chain_length_internal, dimension_internal, num_segments_internal);
		copy_data_to_device(&plans[i], chains_internal, chain_length_internal, dimension_internal, num_segments_internal);
	}
	plans_global = plans;

	//ThreadPoolKernelLaunch kernelpool;
	{
		threadPool<> kernelpool(device_num, ac_gpu_wrapper);
		for(int i =0; i< dimension_internal*num_segments_internal; i++){
			kernelpool.enqueue(i);
		}
	}
	//Wait for final jobs to finish before deallocating gpu memory
	for(int i = 0 ; i< device_num; i++){
		hipSetDevice(i);
		hipStreamSynchronize(plans_global[i].stream);
	}

	//Copy over data from Device to Host
	double **lags = allocate_2D_array(device_num, num_segments_internal*dimension_internal);
	int *lags_transfer;
	hipHostMalloc((void **)&lags_transfer, sizeof(int)* num_segments_internal*dimension_internal);
	for(int i =0; i<device_num; i++){
		hipSetDevice(i);
		hipMemcpyAsync(lags_transfer, plans_global[i].device_lags, 
			sizeof(int)*dimension_internal*num_segments_internal, 
			hipMemcpyDeviceToHost,plans_global[i].stream );
		hipStreamSynchronize(plans_global[i].stream);
		for(int j =0; j<num_segments_internal*dimension_internal; j++)
			lags[i][j] = lags_transfer[j];
	}
	hipHostFree(lags_transfer);
	for(int i =0 ; i<num_segments_internal*dimension_internal; i++)
	{
		for(int j = 0; j<device_num; j++){
			if(lags[j][i] != 2*chain_length_internal){
				int dim = i/num_segments_internal;
				int k = i - dim*num_segments_internal;
				autocorr[dim][k] = lags[j][i];
			}
		}
	}
	deallocate_2D_array(lags, device_num, num_segments_internal);
		
	std::cout<<"STREAMS SYNCED"<<std::endl;
	std::cout<<"DEALLOCATING"<<std::endl;
	for(int i = 0 ; i< device_num; i++)
	{
		deallocate_gpu_plan(&plans_global[i], chain_length_internal, dimension_internal, num_segments);	
	}
	
	for(int i =0 ; i< device_num; i++){
		hipSetDevice(i);
		hipDeviceReset();
	}
	std::cout<<std::endl;

	
}
/*! \brief Wrapper function for the thread pool
 */
void ac_gpu_wrapper(int thread, /**< Host thread*/
			int job_id/**< Job ID*/
			)
{
	launch_ac_gpu(thread, job_id, chains_internal, 
		chain_length_internal, dimension_internal, 
		target_corr_internal, num_segments_internal);
}

/*! \brief Launch the GPU kernel, formatted for the thread pool
 */
void launch_ac_gpu(int device, int element, double **data, int length, int dimension, double target_corr, int num_segments)
{
	//std::cout<<"Thread (device): "<<device<<std::endl;
	hipSetDevice(device);
	int dim = element/num_segments;
	int k = element-dim*num_segments;
	int length_step = length / num_segments;
	int length_seg = (k+1) * length_step;
	int *host_seg;
	int start_id = dim * length;
	//plans_global[device].initial_lag = &length_seg;

	double sum = 0;
	for (int i =start_id ; i< start_id + length_seg; i++){
		sum+=plans_global[device].host_data[i];
	}
	double average = sum/length_seg;

	double var=0;
	auto_corr_internal( plans_global[device].host_data, 
				length_seg, 0, average, &var, start_id);

	//hipMemcpy(plans_global[device].device_data, 
	//		chains_internal, 
	//		sizeof(double)*length, 
	//		hipMemcpyHostToDevice);
	//hipMemcpy(&plans_global[device].device_lags[element], 
	//		&length_seg,
	//		sizeof(int), hipMemcpyHostToDevice
	//		);
	//hipMemcpyAsync(plans_global[device].device_data, 
	//		plans_global[device].host_data, 
	//		sizeof(double)*length_seg, 
	//		hipMemcpyHostToDevice, plans_global[device].stream);
	//hipMemcpyAsync(&plans_global[device].device_lags[element], 
	//		plans_global[device].initial_lag,
	//		sizeof(int), hipMemcpyHostToDevice, 
	//		plans_global[device].stream);
	
	int N = length_seg;
	auto_corr_internal_kernal
		<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK, 
		0, plans_global[device].stream>>>
		(plans_global[device].device_data, length_seg, average, 
		&plans_global[device].device_lags[element], target_corr, var, start_id);

	//auto_corr_internal_kernal
	//	<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>
	//	(plans_global[device].device_data, length_seg, average, 
	//	&plans_global[device].device_lags[element], target_corr, var, start_id);
	//hipMemcpy(plans_global[device].host_lag, 
	//		&plans_global[device].device_lags[element], sizeof(int), 
	//		hipMemcpyDeviceToHost);
	hipStreamSynchronize(plans_global[device].stream);
	//hipDeviceSynchronize();
	//return *plans_global[device].host_lag - start_id;
	//std::cout<<element<<std::endl;
	//return 1;//*plans_global[device].host_lag ;
}

/*! \brief Allocates memory for autocorrelation--GPU structure
 */
void allocate_gpu_plan(GPUplan *plan, /**< Structure for GPU plan*/
		int data_length, /**< Length of data*/
		int dimension, /**< Dimension of the data*/
		int num_segments /**< Number of segments to calculate the autocorrelation length*/
		)
{
	hipSetDevice(plan->device_id);
	
	hipMalloc((void **)&plan->device_data, sizeof(double)*data_length*dimension);
	hipHostMalloc((void **)&plan->host_data, sizeof(double)*data_length*dimension);
	hipMalloc((void **)&plan->device_lag, sizeof(int));
	hipMalloc((void **)&plan->device_lags, sizeof(int)*dimension*num_segments);
	hipHostMalloc((void **)&plan->host_lag, sizeof(int));
	hipHostMalloc((void **)&plan->initial_lag, sizeof(int));
	hipStreamCreate(&plan->stream);
}
/*! \brief Deallocates memory for the autocorrelation--GPU structure 
 */
void deallocate_gpu_plan(GPUplan *plan, /**< Structure for the GPU plan*/
		int data_length, /**< Length of data*/
		int dimension, /**< Dimension of the data*/
		int num_segments /**< Number of segments to calculate the autocorrelation length*/
		)
{	
	hipSetDevice(plan->device_id);
	hipFree(plan->device_data);
	hipFree(plan->device_lag);
	hipFree(plan->device_lags);
	hipHostFree(plan->host_lag);
	hipHostFree(plan->initial_lag);
	hipHostFree(plan->host_data);
	hipStreamDestroy(plan->stream);
}
/*! \brief Copy data to device before starting kernels
 */
void copy_data_to_device(GPUplan *plan, /**< GPU plan*/
		double **input_data, /**<Input chain data*/
		int data_length, /**< Length of data*/
		int dimension, /**< Dimension of the data*/
		int num_segments /**< Number of segments to calculate the autocorrelation length*/
		)
{
	hipSetDevice(plan->device_id);
	for(int i =0; i< dimension; i++){
		for(int j =0; j<data_length; j++){
			plan->host_data[i*data_length + j] = input_data[j][i];
		}
	}
	hipMemcpyAsync(plan->device_data, plan->host_data, 
		sizeof(double)*data_length*dimension, 
		hipMemcpyHostToDevice,plan->stream );
	int * data_lengths;
	hipHostMalloc((void **)&data_lengths, sizeof(int)*num_segments*dimension);
	for(int i =0 ;i < num_segments*dimension; i++)
		data_lengths[i] = data_length*2;
	hipMemcpyAsync(plan->device_lags, data_lengths, 
		sizeof(int)*dimension*num_segments, 
		hipMemcpyHostToDevice,plan->stream );
	hipHostFree(data_lengths);
	
}



/*! \brief Faster approximation of the autocorrelation of a chain. Implements FFT/IFFT -- GPU accelerated
 *
 * Based on the Wiener-Khinchin Theorem.
 *
 * Algorithm used from https://lingpipe-blog.com/2012/06/08/autocorrelation-fft-kiss-eigen/
 *
 */
//void auto_correlation_spectral_accel(double *chain, int length, double *autocorr)
//{
//
//	//Normalize
//	double *x_cent = (double *)malloc(sizeof(double)*length);
//	//Calculate Average
//	double ave = 0;
//	for(int i =0; i<length; i++)
//		ave+= chain[i];
//	ave /= length;
//	
//	//Create normalized vector
//	for(int i = 0 ; i<length; i++){
//		x_cent[i] = chain[i]-ave;
//	}
//
//	//Padded length
//	int L = pow(2, std::ceil( std::log2(length) ) );	
//
//	//Padded Vector
//	double *x_pad = (double *)malloc(sizeof(double)*L);
//
//	//Copy centered vector
//	for(int i = 0 ; i < length; i++){
//		x_pad[i] = x_cent[i];
//	}
//
//	//Add padding
//	for(int i = length ; i < L; i++){
//		x_pad[i] = 0;
//	}
//
//	//Allocate FFTW3 memory
//	double *norm = (double *)malloc(sizeof(double)*L);
//	std::complex<double> *out_host = (std::complex<double> *)malloc(sizeof(std::complex<double>)*L);
//	//fftw_outline plan;
//	//hipfftHandle plan; 
//	hipfftReal *in;
//	hipfftComplex *out;
//	int batch = 1;
//	
//	hipMalloc((void **) &in, sizeof(cfftReal)*L*batch);
//	hipMalloc((void **) &out, sizeof(cfftComplex)*L*batch);
//	hipfftPlan1d(&plan, L, CUFFT D2Z, batch); 
//	hipfftExecD2Z(plan, in, data, CUFFT FORWARD);
//	hipMemcpy(out_host,out, sizeof(std::complex<double>)*L, hipMemcpyDeviceToHost);
//	hipfftDestroy(plan); 
//	hipFree(in);
//	hipFree(out);
//	//initiate_likelihood_function(&plan, L);
//	//
//	//fftw_complex *in = (fftw_complex*)fftw_malloc(sizeof(fftw_complex)*L);
//	//fftw_complex *out = (fftw_complex*)fftw_malloc(sizeof(fftw_complex)*L);
//	//for(int i =0 ; i<L; i++){
//	//	in[i][0] = x_pad[i];
//	//	in[i][1] = 0;
//	//}
//
//	////Execute Forward Transform
//	//fftw_execute_dft(plan.p, in, out);
//
//	//Take norm^2 of the output
//	for(int i =0 ; i<L; i++){
//		double re = std::real(out_host[i]);
//		double im = std::imag(out_host[i]);
//		norm[i] = re*re + im*im;
//	}
//	//Execute Reverse Transform
//	fftw_outline plan_inv;
//	allocate_FFTW3_mem_inverse(&plan_inv, L);
//	for(int i =0 ; i<L; i++){
//		in[i][0] = norm[i];
//		in[i][1] = 0;
//	}
//	fftw_execute_dft(plan_inv.p, in, out);
//	
//	//acov is the result
//	double *acov = (double *)malloc(sizeof(double)*length);
//	for(int i =0 ; i< length; i++){
//		acov[i] = out[i][0];	
//	}
//
//	//adjust the cov
//	double *mask = (double *)malloc(sizeof(double)*L);
//	//first length elements are 1
//	for(int i = 0 ; i < length; i++){
//		mask[i]=1;
//	}
//	// last L-length elements are 0
//	for(int i = length ; i < L; i++){
//		mask[i]=0;
//	}
//	for(int i =0 ; i<L; i++){
//		in[i][0] = mask[i];
//		in[i][1] = 0;
//	}
//
//	//execute fft
//	fftw_execute_dft(plan.p, in ,out);
//	
//	//output vector -- will be trimmed to length
//	double *normadj = (double *)malloc(sizeof(double)*length);
//	//trimmed output
//	for(int i =0 ; i< length; i++){
//		normadj[i] = out[i][0]*out[i][0] + out[i][1]*out[i][1];
//	}
//	
//	//adjust the cov vector
//	//for(int i =0 ; i<length ; i++){
//	//	acov[i]/=normadj[i];
//	//}
//	
//	double var = acov[0];
//
//	for(int i = 0 ; i< length; i++)
//		autocorr[i] = acov[i]/var;
//
//	//Free memory
//	deactivate_likelihood_function(&plan);
//	deactivate_likelihood_function(&plan_inv);
//	free(norm);
//	free(out_host);
//	free(mask);
//	free(normadj);
//	fftw_free(in);
//	fftw_free(out);
//	free(x_cent);
//	free(x_pad);
//}
