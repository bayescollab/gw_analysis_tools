#include "hip/hip_runtime.h"
#include "cuda_utilities.h"
#include "cuda_utilities.hu"
#include "util.h"
#include <iostream>
#include <condition_variable>
#include <thread>
#include <queue>
#include <functional>
#include <mutex>
#include <unistd.h>

/*! \file
 */
GPUplan *plans_global;
class ThreadPoolKernelLaunch
{
public:
	explicit ThreadPoolKernelLaunch()
	{
		int device_num;
		hipGetDeviceCount(&device_num);
		start(device_num);
		//start(1);
	}

	~ThreadPoolKernelLaunch()
	{
		stop();
	}


	void enqueue(int i)
	{
		{
			std::unique_lock<std::mutex> lock{mEventMutex};
			mTasks.emplace(std::move(i));
		}
		mEventVar.notify_one();
	}

	void public_stop()
	{
		stop();
	}
	int get_queue_length()
	{
		return mTasks.size();
	}
	void end_pool()
	{
		while(true)
		{
			if(mTasks.empty()){
				stop();
			}
			usleep(100);	
		}
	}
private:
	
	std::vector<std::thread> mThreads;
	std::condition_variable mEventVar;
	std::mutex mEventMutex;
	bool mStopping = false;
	std::queue<int> mTasks;

	void start(std::size_t numThreads)
	{
		for(auto i =0u; i<numThreads; i++)
		{
			mThreads.emplace_back([=]{
				while(true)
				{
					int j;
					{
						std::unique_lock<std::mutex> lock{mEventMutex};
						mEventVar.wait(lock,[=]{return mStopping || !mTasks.empty(); });
						
						if (mStopping && mTasks.empty())
							break;	
						j = std::move(mTasks.front());
						mTasks.pop();
					}
					//std::cout<<"DEVICE: "<<i<<std::endl;
					ac_gpu_wrapper(i, j);
					
				}
			});
		}

	}
	void stop() noexcept
	{
		std::cout<<std::endl;
		std::cout<<"Stop initiated -- waiting for threads to finish"<<std::endl;
		{
			std::unique_lock<std::mutex> lock{mEventMutex};
			mStopping = true;
		}
		
		mEventVar.notify_all();
		
		for(auto &thread: mThreads)
			thread.join();
	}
	
};

__device__ __host__
void auto_corr_internal(double *arr, int length, int lag, double average, double *corr, int start_id)
{
	double sum = 0;
	for(int i =0; i< (length - lag); i++){
		sum+= (arr[i+lag+start_id] - average ) * ( arr[i+start_id] - average );
	}		
	*corr = sum / (length - lag);
}
__global__ 
void auto_corr_internal_prep(int *lag, int length)
{
	*lag = length;
}

__global__
void auto_corr_internal_kernal(double *arr, int length,  double average, int *rho_index, double target_corr, double var, int start_id)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id < *rho_index){
		double corr;
		auto_corr_internal(arr, length, id, average, &corr, start_id);
		if(corr/var<target_corr) atomicMin(rho_index, id);
	}

}

void auto_corr_from_data_accel(double **output, int dimension, int N_steps, int num_segments, double target_corr, double **autocorr)
{
	int device_num;
	//int current_dev;
	//int succ;
	hipGetDeviceCount(&device_num);
	//std::cout<<"NUMBER OF DEVICES "<<device_num<<std::endl;
	//hipGetDevice(&current_dev);
	//std::cout<<"CURRENT DEV: "<<current_dev<<std::endl;
	//succ = hipSetDevice(0);
	//std::cout<<"SUCCESS: "<<succ<<std::endl;
	//hipGetDevice(&current_dev);
	//std::cout<<"CURRENT DEV: "<<current_dev<<std::endl;

	if(device_num==1){
	//if(true){
		int *rho_index;
		//HERE
		hipMallocManaged( (void**)&rho_index, sizeof(int) );
		int dim ;
		int length_step = N_steps / num_segments;
		int iterations = dimension * num_segments;
		for(dim=0; dim<dimension; dim ++){
			for(int k =0 ; k<num_segments; k++){
				//std::cout<<"DIM: "<<dim<<std::endl;
				//std::cout<<"k: "<<k<<std::endl;
				//std::cout<<"LOOP: "<<dim*num_segments + k<<std::endl;
				int length_seg = (k+1) * length_step;
				int laginit = length_seg;

				double *temp = (double*) malloc(sizeof(double)* length_seg);
				double *arr;
				//HERE
				hipMallocManaged( (void**)&arr, length_seg*sizeof(double) );

				for(int	j = 0 ; j< length_seg; j++){
					temp[j] = output[j][dim];	
				}

				double sum = 0;
				for (int i =0 ; i< length_seg; i++){
					sum+=temp[i];
				}
				double average = sum/length_seg;

				double var=0;
				auto_corr_internal( temp, length_seg, 0, average, &var, 0);

				hipMemcpy(arr, temp, sizeof(double)*length_seg, hipMemcpyHostToDevice);
				hipMemcpy(rho_index, &laginit, sizeof(int), hipMemcpyHostToDevice);
				
				int N = length_seg;
				
				//std::cout<<"LAUNCHING KERNAL"<<std::endl;
				auto_corr_internal_kernal
					<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>
					(arr, length_seg, average, rho_index, target_corr, var, 0);

				int lag ;
				hipMemcpy(&lag, rho_index, sizeof(int), hipMemcpyDeviceToHost);
				//hipDeviceSynchronize();
				//std::cout<<"COPYING RESULTS"<<std::endl;
				autocorr[k][dim] = lag;
				free(temp);
				hipFree(arr);
				printProgress((double)(dim*num_segments + k)/iterations);
				
			}
		}
		hipFree(rho_index);
	}
	else{
		chains_internal = output;
		target_corr_internal = target_corr;
		chain_length_internal = N_steps;
		num_segments_internal = num_segments;
		dimension_internal = dimension;
		autocorr_internal = autocorr;

		//GPUplan plans_local[gpu_count];
		//plans = plans_local;
		GPUplan plans[device_num];
		
		for(int i = 0 ; i< device_num; i++)
		{
			plans[i].device_id = i;
			allocate_gpu_plan(&plans[i],chain_length_internal, dimension_internal, num_segments_internal);
			copy_data_to_device(&plans[i], chains_internal, chain_length_internal, dimension_internal, num_segments_internal);
		}
		plans_global = plans;

		//ThreadPoolKernelLaunch kernelpool(output, dimension, N_steps, autocorr, target_corr, num_segments);
		ThreadPoolKernelLaunch kernelpool;
		for(int i =0; i< dimension_internal*num_segments_internal; i++){
			kernelpool.enqueue(i);
		}
		//Wait for queue to empty before deallocating gpu memory
		while(kernelpool.get_queue_length() != 0){
			usleep(50000);
		}	
		//Wait for final jobs to finish before deallocating gpu memory
		for(int i = 0 ; i< device_num; i++){
			hipSetDevice(i);
			hipStreamSynchronize(plans_global[i].stream);
		}
		double **lags = allocate_2D_array(device_num, num_segments_internal*dimension_internal);
		int *lags_transfer;
		hipHostMalloc((void **)&lags_transfer, sizeof(int)* num_segments_internal*dimension_internal);
		for(int i =0; i<device_num; i++){
			hipSetDevice(i);
			hipMemcpyAsync(lags_transfer, plans_global[i].device_lags, 
				sizeof(int)*dimension_internal*num_segments_internal, 
				hipMemcpyDeviceToHost,plans_global[i].stream );
			hipStreamSynchronize(plans_global[i].stream);
			for(int j =0; j<num_segments_internal*dimension_internal; j++)
				//std::cout<<lags_transfer[j]<<std::endl;
				lags[i][j] = lags_transfer[j];
		}
		hipHostFree(lags_transfer);
		//Wait for final jobs to finish before deallocating gpu memory
		for(int i = 0 ; i< device_num; i++){
			hipSetDevice(i);
			hipStreamSynchronize(plans_global[i].stream);
		}
		for(int i =0 ; i<num_segments_internal*dimension_internal; i++)
		{
			for(int j = 0; j<device_num; j++){
				if(lags[j][i] != 2*chain_length_internal){
					int dim = i/num_segments_internal;
					int k = i - dim*num_segments_internal;
					autocorr[k][dim] = lags[j][i];
				}
			}
		}
		deallocate_2D_array(lags, device_num, num_segments_internal);
			
		std::cout<<"STREAMS SYNCED"<<std::endl;
		std::cout<<"DEALLOCATING"<<std::endl;
		for(int i = 0 ; i< device_num; i++)
		{
			deallocate_gpu_plan(&plans_global[i], chain_length_internal, dimension_internal, num_segments);	
		}
	}
	for(int i = 0; i<dimension; i++){
		for(int j =0; j<num_segments; j++){
			std::cout<<i<<" "<<j<<" "<<autocorr[j][i]<<std::endl;
		}
	}
	for(int i =0 ; i< device_num; i++){
		hipSetDevice(i);
		hipDeviceReset();
	}
	std::cout<<std::endl;

	
}
void ac_gpu_wrapper(int thread, int job_id)
{
	int dim = job_id/num_segments_internal;
	int k = job_id-dim*num_segments_internal;
	autocorr_internal[k][dim] = 
		launch_ac_gpu(thread, job_id, chains_internal, 
		chain_length_internal, dimension_internal, 
		target_corr_internal, num_segments_internal);
}

int launch_ac_gpu(int device, int element, double **data, int length, int dimension, double target_corr, int num_segments)
{
	hipSetDevice(device);
	int dim = element/num_segments;
	int k = element-dim*num_segments;
	int length_step = length / num_segments;
	int length_seg = (k+1) * length_step;
	int *host_seg;
	int start_id = dim * length;
	plans_global[device].initial_lag = &length_seg;

	double sum = 0;
	for (int i =start_id ; i< start_id + length_seg; i++){
		sum+=plans_global[device].host_data[i];
	}
	double average = sum/length_seg;

	double var=0;
	auto_corr_internal( plans_global[device].host_data, 
				length_seg, 0, average, &var, start_id);

	//hipMemcpyAsync(plans_global[device].device_data, 
	//		plans_global[device].host_data, 
	//		sizeof(double)*length_seg, 
	//		hipMemcpyHostToDevice, 
	//		plans_global[device].stream);
	//hipMemcpyAsync(plans_global[device].device_lag, 
	//		plans_global[device].initial_lag,
	//		sizeof(int), hipMemcpyHostToDevice, 
	//		plans_global[device].stream);
	//printf("%d\n", *plans_global[device].device_lag);
	//std::cout<<*plans_global[device].initial_lag<<std::endl;
	
	int N = length_seg;
	auto_corr_internal_kernal
		<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK, 
		0, plans_global[device].stream>>>
		(plans_global[device].device_data, length_seg, average, 
		&plans_global[device].device_lags[element], target_corr, var, start_id);

	//printf("%d\n", plans_global[device].device_lag);
	//hipMemcpy(plans_global[device].host_lag, 
	//		&plans_global[device].device_lags[element], sizeof(int), 
	//		hipMemcpyDeviceToHost);
	//hipStreamSynchronize(plans_global[device].stream);
	//return *plans_global[device].host_lag - start_id;
	//std::cout<<element<<std::endl;
	return 1;//*plans_global[device].host_lag ;
}

void allocate_gpu_plan(GPUplan *plan, int data_length, int dimension, int num_segments)
{
	hipSetDevice(plan->device_id);
	
	hipMalloc((void **)&plan->device_data, sizeof(double)*data_length*dimension);
	hipHostMalloc((void **)&plan->host_data, sizeof(double)*data_length*dimension);
	hipMalloc((void **)&plan->device_lag, sizeof(int));
	hipMalloc((void **)&plan->device_lags, sizeof(int)*dimension*num_segments);
	hipHostMalloc((void **)&plan->host_lag, sizeof(int));
	hipHostMalloc((void **)&plan->initial_lag, sizeof(int));
	hipStreamCreate(&plan->stream);
}
void deallocate_gpu_plan(GPUplan *plan, int data_length, int dimension, int num_segments)
{	
	hipSetDevice(plan->device_id);
	hipFree(plan->device_data);
	hipFree(plan->device_lag);
	hipFree(plan->device_lags);
	hipHostFree(plan->host_lag);
	hipHostFree(plan->initial_lag);
	hipHostFree(plan->host_data);
	hipStreamDestroy(plan->stream);
}
void copy_data_to_device(GPUplan *plan, double **input_data,int data_length, int dimension, int num_segments)
{
	hipSetDevice(plan->device_id);
	for(int i =0; i< dimension; i++){
		for(int j =0; j<data_length; j++){
			plan->host_data[i*data_length + j] = input_data[j][i];
		}
	}
	hipMemcpyAsync(plan->device_data, plan->host_data, 
		sizeof(double)*data_length*dimension, 
		hipMemcpyHostToDevice,plan->stream );
	int * data_lengths;
	hipHostMalloc((void **)&data_lengths, sizeof(int)*num_segments*dimension);
	for(int i =0 ;i < num_segments*dimension; i++)
		data_lengths[i] = data_length*2;
	hipMemcpyAsync(plan->device_lags, data_lengths, 
		sizeof(int)*dimension*num_segments, 
		hipMemcpyHostToDevice,plan->stream );
	hipHostFree(data_lengths);
	
}
