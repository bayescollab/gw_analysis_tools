#include "hip/hip_runtime.h"
#include "cuda_utilities.h"
#include "cuda_utilities.hu"
#include "util.h"
#include <iostream>

__device__ __host__
void auto_corr_internal(double *arr, int length, int lag, double average, double *corr)
{
	double sum = 0;
	for(int i =0; i< (length - lag); i++){
		sum+= (arr[i+lag] - average ) * ( arr[i] - average );
	}		
	*corr = sum / (length - lag);
}
__global__
void auto_corr_internal_kernal(double *arr, int length,  double average, int *rho_index, double target_corr, double var)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id < *rho_index){
		double corr;
		auto_corr_internal(arr, length, id, average, &corr);
		if(corr/var<target_corr) atomicMin(rho_index, id);
	}

}

void auto_corr_from_data_accel(double **output, int dimension, int N_steps, int num_segments, double target_corr, double **autocorr)
{

	int *rho_index;

	hipMalloc( (void**)&rho_index, sizeof(int) );

	//double target_corr = .01;

	int dim ;
	int length_step = N_steps / num_segments;
	int threads_per_block = 512;
	int iterations = dimension * num_segments;

	for(dim=0; dim<dimension; dim ++){
		for(int k =0 ; k<num_segments; k++){
			int length_seg = (k+1) * length_step;
			int laginit = length_seg;

			double *temp = (double*) malloc(sizeof(double)* length_seg);
			double *arr;
			hipMalloc( (void**)&arr, length_seg*sizeof(double) );

			for(int	j = 0 ; j< length_seg; j++){
				temp[j] = output[j][dim];	
			}

			double sum = 0;
			for (int i =0 ; i< length_seg; i++){
				sum+=temp[i];
			}
			double average = sum/length_seg;

			double var=0;
			auto_corr_internal( temp, length_seg, 0, average, &var);

			hipMemcpy(arr, temp, sizeof(double)*length_seg, hipMemcpyHostToDevice);
			hipMemcpy(rho_index, &laginit, sizeof(int), hipMemcpyHostToDevice);
			
			int N = length_seg;
			
			auto_corr_internal_kernal
				<<<N/threads_per_block,threads_per_block>>>
				(arr, length_seg, average, rho_index, target_corr, var);

			int lag ;
			hipMemcpy(&lag, rho_index, sizeof(int), hipMemcpyDeviceToHost);
			autocorr[k][dim] = lag;
			free(temp);
			hipFree(arr);
			printProgress((double)(dim*num_segments + k)/iterations);
			
		}
	}
	std::cout<<std::endl;
	hipFree(rho_index);

	
}
