#include "hip/hip_runtime.h"
#include "cuda_utilities.h"
#include "cuda_utilities.hu"
#include "util.h"
#include <iostream>
#include <condition_variable>
#include <thread>
#include <queue>
#include <functional>
#include <mutex>
#include <unistd.h>
#include <threadPool.h>

/*! \file
 */

GPUplan *plans_global;

/*! \brief Internal function to calculate the autocorrelation for a given lag
 * Customized for the thread pool architecture, with extra arguments because of the way the memory is allocated
 */
__device__ __host__
void auto_corr_internal(double *arr, /**< Input array of data*/
			int length, /**< Length of input array*/
			int lag,  /**< Lag to be used to calculate the correlation*/
			double average,  /**< Average of the array arr*/
			double *corr,  /**< [out] output correlation*/
			int start_id /**< ID of location to start calculation -- input arrary arr is assumed to be contiguous for multiple dimensions*/
			)
{
	double sum = 0;
	for(int i =0; i< (length - lag); i++){
		sum+= (arr[i+lag+start_id] - average ) * ( arr[i+start_id] - average );
	}		
	*corr = sum / (length - lag);
}

/*! \brief Internal function to launch the CUDA kernel for a range of autocorrelations
 * 
 * Correlation function used:
 *
 * rho(lag) = 1 / (length - lag) \sum (arr[i+lag]-average) ( arr[i]- average)
 *
 * target_corr = rho(rho_index)/rho(0) = rho(rho_index)/var
 */
__global__
void auto_corr_internal_kernal(double *arr, /**< Input array of data*/
				int length,  /**< Length of data array*/
				double average, /**< Average of input data*/
				int *rho_index, /**< [out] Index of the lag that results ina correlation ratio target_corr*/
				double target_corr, /**< Target correlation ratio rho(lag)/rho(0) = target_corr*/
				double var, /**< Variance rho(0)*/
				int start_id/**< Starting index to use for the data array arr*/
				)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id < *rho_index){
		double corr;
		auto_corr_internal(arr, length, id, average, &corr, start_id);
		if(corr/var<target_corr) atomicMin(rho_index, id);
	}

}

/*! \brief Write data file for autocorrelation lengths of the data given a data file name, as written by the mcmc_sampler
 */
void write_file_auto_corr_from_data_file_accel(std::string acfile, /**< Filename of the autocorrelation data*/
					std::string chains_file, /**<Filename of the data file for the chains*/
					int dimension, /**< Dimension of the data*/
					int N_steps, /**< Number of steps in the chain*/
					int num_segments,  /**< Number of segments to check the autocorrelation length for each dimension*/
					double target_corr/**< Target correlation ratio to use for the correlation length calculation*/
					)
{
	double **chains = allocate_2D_array(N_steps, dimension);
	read_file(chains_file, chains, N_steps, dimension);
	write_file_auto_corr_from_data_accel(acfile, chains, dimension, 
			N_steps, num_segments, target_corr);	
	deallocate_2D_array(chains,N_steps, dimension);
}

/*! \brief Write data file given output chains, as formatted by the mcmc_sampler
 */
void write_file_auto_corr_from_data_accel(std::string acfile, /**< Output autocorrelation filename */
					double **chains, /**< Chain data from MCMC_sampler*/
					int dimension, /**< Dimension of the data*/
					int N_steps, /**< Number of steps in the chain*/
					int num_segments,  /**< Number of segments to check the autocorrelation length for each dimension*/
					double target_corr/**< Target correlation ratio to use for the correlation length calculation*/
					)
{
	double **autocorr = allocate_2D_array(dimension, num_segments);
	double **autocorrout = allocate_2D_array(dimension+1, num_segments);
	auto_corr_from_data_accel(chains, dimension, N_steps, num_segments, target_corr, autocorr);
	int seg_step = N_steps/ num_segments;
	for(int i =0 ; i < num_segments; i ++)
	{
		autocorrout[0][i] = (i+1) * seg_step;
		for(int j =0; j<dimension; j++){
			autocorrout[j+1][i] = autocorr[j][i];
		}
	}
	write_file(acfile, autocorrout, dimension+1, num_segments);
}

/*! \brief Find autocorrelation of data at different points in the chain length and output to autocorr
 */
void auto_corr_from_data_accel(double **output, /**< Chain data input*/
				int dimension, /**< Dimension of the data*/
				int N_steps, /**< Number of steps in the data*/
				int num_segments, /**< number of segments to calculate the autocorrelation length*/
				double target_corr, /**< Target correlation ratio*/
				double **autocorr /**<[out] Autocorrelation lengths for the different segments*/
				)
{
	int device_num;
	hipGetDeviceCount(&device_num);
	chains_internal = output;
	target_corr_internal = target_corr;
	chain_length_internal = N_steps;
	num_segments_internal = num_segments;
	dimension_internal = dimension;
	autocorr_internal = autocorr;

	GPUplan plans[device_num];
	
	for(int i = 0 ; i< device_num; i++)
	{
		plans[i].device_id = i;
		allocate_gpu_plan(&plans[i],chain_length_internal, dimension_internal, num_segments_internal);
		copy_data_to_device(&plans[i], chains_internal, chain_length_internal, dimension_internal, num_segments_internal);
	}
	plans_global = plans;

	//ThreadPoolKernelLaunch kernelpool;
	{
		threadPool<> kernelpool(device_num, ac_gpu_wrapper);
		for(int i =0; i< dimension_internal*num_segments_internal; i++){
			kernelpool.enqueue(i);
		}
	}
	//Wait for final jobs to finish before deallocating gpu memory
	for(int i = 0 ; i< device_num; i++){
		hipSetDevice(i);
		hipStreamSynchronize(plans_global[i].stream);
	}

	//Copy over data from Device to Host
	double **lags = allocate_2D_array(device_num, num_segments_internal*dimension_internal);
	int *lags_transfer;
	hipHostMalloc((void **)&lags_transfer, sizeof(int)* num_segments_internal*dimension_internal);
	for(int i =0; i<device_num; i++){
		hipSetDevice(i);
		hipMemcpyAsync(lags_transfer, plans_global[i].device_lags, 
			sizeof(int)*dimension_internal*num_segments_internal, 
			hipMemcpyDeviceToHost,plans_global[i].stream );
		hipStreamSynchronize(plans_global[i].stream);
		for(int j =0; j<num_segments_internal*dimension_internal; j++)
			lags[i][j] = lags_transfer[j];
	}
	hipHostFree(lags_transfer);
	for(int i =0 ; i<num_segments_internal*dimension_internal; i++)
	{
		for(int j = 0; j<device_num; j++){
			if(lags[j][i] != 2*chain_length_internal){
				int dim = i/num_segments_internal;
				int k = i - dim*num_segments_internal;
				autocorr[dim][k] = lags[j][i];
			}
		}
	}
	deallocate_2D_array(lags, device_num, num_segments_internal);
		
	std::cout<<"STREAMS SYNCED"<<std::endl;
	std::cout<<"DEALLOCATING"<<std::endl;
	for(int i = 0 ; i< device_num; i++)
	{
		deallocate_gpu_plan(&plans_global[i], chain_length_internal, dimension_internal, num_segments);	
	}
	
	for(int i =0 ; i< device_num; i++){
		hipSetDevice(i);
		hipDeviceReset();
	}
	std::cout<<std::endl;

	
}
/*! \brief Wrapper function for the thread pool
 */
void ac_gpu_wrapper(int thread, /**< Host thread*/
			int job_id/**< Job ID*/
			)
{
	launch_ac_gpu(thread, job_id, chains_internal, 
		chain_length_internal, dimension_internal, 
		target_corr_internal, num_segments_internal);
}

/*! \brief Launch the GPU kernel, formatted for the thread pool
 */
void launch_ac_gpu(int device, int element, double **data, int length, int dimension, double target_corr, int num_segments)
{
	hipSetDevice(device);
	int dim = element/num_segments;
	int k = element-dim*num_segments;
	int length_step = length / num_segments;
	int length_seg = (k+1) * length_step;
	int *host_seg;
	int start_id = dim * length;
	//plans_global[device].initial_lag = &length_seg;

	double sum = 0;
	for (int i =start_id ; i< start_id + length_seg; i++){
		sum+=plans_global[device].host_data[i];
	}
	double average = sum/length_seg;

	double var=0;
	auto_corr_internal( plans_global[device].host_data, 
				length_seg, 0, average, &var, start_id);

	//hipMemcpyAsync(plans_global[device].device_data, 
	//		plans_global[device].host_data, 
	//		sizeof(double)*length_seg, 
	//		hipMemcpyHostToDevice, 
	//		plans_global[device].stream);
	//hipMemcpyAsync(&plans_global[device].device_lags[element], 
	//		plans_global[device].initial_lag,
	//		sizeof(int), hipMemcpyHostToDevice, 
	//		plans_global[device].stream);
	
	int N = length_seg;
	auto_corr_internal_kernal
		<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK, 
		0, plans_global[device].stream>>>
		(plans_global[device].device_data, length_seg, average, 
		&plans_global[device].device_lags[element], target_corr, var, start_id);

	//hipMemcpy(plans_global[device].host_lag, 
	//		&plans_global[device].device_lags[element], sizeof(int), 
	//		hipMemcpyDeviceToHost);
	//hipStreamSynchronize(plans_global[device].stream);
	//return *plans_global[device].host_lag - start_id;
	//std::cout<<element<<std::endl;
	//return 1;//*plans_global[device].host_lag ;
}

/*! \brief Allocates memory for autocorrelation--GPU structure
 */
void allocate_gpu_plan(GPUplan *plan, /**< Structure for GPU plan*/
		int data_length, /**< Length of data*/
		int dimension, /**< Dimension of the data*/
		int num_segments /**< Number of segments to calculate the autocorrelation length*/
		)
{
	hipSetDevice(plan->device_id);
	
	hipMalloc((void **)&plan->device_data, sizeof(double)*data_length*dimension);
	hipHostMalloc((void **)&plan->host_data, sizeof(double)*data_length*dimension);
	hipMalloc((void **)&plan->device_lag, sizeof(int));
	hipMalloc((void **)&plan->device_lags, sizeof(int)*dimension*num_segments);
	hipHostMalloc((void **)&plan->host_lag, sizeof(int));
	hipHostMalloc((void **)&plan->initial_lag, sizeof(int));
	hipStreamCreate(&plan->stream);
}
/*! \brief Deallocates memory for the autocorrelation--GPU structure 
 */
void deallocate_gpu_plan(GPUplan *plan, /**< Structure for the GPU plan*/
		int data_length, /**< Length of data*/
		int dimension, /**< Dimension of the data*/
		int num_segments /**< Number of segments to calculate the autocorrelation length*/
		)
{	
	hipSetDevice(plan->device_id);
	hipFree(plan->device_data);
	hipFree(plan->device_lag);
	hipFree(plan->device_lags);
	hipHostFree(plan->host_lag);
	hipHostFree(plan->initial_lag);
	hipHostFree(plan->host_data);
	hipStreamDestroy(plan->stream);
}
/*! \brief Copy data to device before starting kernels
 */
void copy_data_to_device(GPUplan *plan, /**< GPU plan*/
		double **input_data, /**<Input chain data*/
		int data_length, /**< Length of data*/
		int dimension, /**< Dimension of the data*/
		int num_segments /**< Number of segments to calculate the autocorrelation length*/
		)
{
	hipSetDevice(plan->device_id);
	for(int i =0; i< dimension; i++){
		for(int j =0; j<data_length; j++){
			plan->host_data[i*data_length + j] = input_data[j][i];
		}
	}
	hipMemcpyAsync(plan->device_data, plan->host_data, 
		sizeof(double)*data_length*dimension, 
		hipMemcpyHostToDevice,plan->stream );
	int * data_lengths;
	hipHostMalloc((void **)&data_lengths, sizeof(int)*num_segments*dimension);
	for(int i =0 ;i < num_segments*dimension; i++)
		data_lengths[i] = data_length*2;
	hipMemcpyAsync(plan->device_lags, data_lengths, 
		sizeof(int)*dimension*num_segments, 
		hipMemcpyHostToDevice,plan->stream );
	hipHostFree(data_lengths);
	
}
