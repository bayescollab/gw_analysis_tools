#include "hip/hip_runtime.h"
#include "cuda_utilities.h"
#include "cuda_utilities.hu"
#include <iostream>

__device__
void auto_corr_internal(double *arr, int length, int lag, double average, double *corr)
{
	double sum = 0;
	for(int i =0; i< (length - lag); i++){
		sum+= (arr[i+lag] - average ) * ( arr[i] - average );
	}		
	*corr = sum / (length - lag);
}
__global__
void auto_corr_internal_kernal(double *arr, int length, int *lag, double average, double *corr)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	//auto_corr_internal(arr, length, lag[id], average, &corr[id]);
	double sum = 0;
	for(int i =0; i< (length - lag[id]); i++){
		sum+= (arr[i+lag[id]] - average ) * ( arr[i] - average );
	}		
	double temp  = sum/ (length - lag[id]);
	//corr[id] = sum / (length - lag[id]);

}

void auto_corr_from_data_accel(double **output, int dimension, int N_steps, double **autocorr)
{
	double *arr, *corr;
	int *lags;
	double average = 0;

	double *temp = (double*) malloc(sizeof(double)* N_steps);
	int *temp2 = (int*) malloc(sizeof(int)* N_steps);
	double *ac = (double*) malloc(sizeof(double)* N_steps);

	hipMalloc( (void**)&arr, N_steps*sizeof(double) );
	hipMalloc( (void**)&corr, N_steps*sizeof(double) );
	hipMalloc( (void**)&lags, N_steps*sizeof(int) );

	int dim = 0;
	for(int	j = 0 ; j< N_steps; j++){
		//std::cout<<output[j][dim]<<std::endl;
		temp[j] = output[j][dim];	
		temp2[j] = j;
	}
	hipMemcpy(arr, temp, sizeof(double)*N_steps, hipMemcpyHostToDevice);
	hipMemcpy(lags, temp2, sizeof(int)*N_steps, hipMemcpyHostToDevice);
	
	int N = 750000;
	int threads_per_block = 512;
	
	auto_corr_internal_kernal<<<(int)((double)N/threads_per_block),threads_per_block>>>(arr, N_steps, lags, average, corr);
	//auto_corr_internal_kernal<<<N_steps,1>>>(arr, N_steps, lags, average, corr);

	//hipMemcpy(ac, corr, sizeof(double)*N_steps, hipMemcpyDeviceToHost);

	for(int i =0; i<100; i++)
		std::cout<<ac[i]<<std::endl;
	hipFree(arr);
	hipFree(corr);

	//int length = N_steps;
	//for(int j =0 ; j<N; j++){
	//	int id = j;
	//	double sum = 0;
	//	for(int i =0; i< (length - temp2[id]); i++){
	//		sum+= (temp[i+temp2[id]] - average ) * ( temp[i] - average );
	//	}		
	//	ac[id] = sum / (length - temp2[id]);
	//}
	free(temp);
	free(temp2);
	free(ac);
	
}
